#include "hip/hip_runtime.h"
/*********************************************************************
This is the source file for the CUDA version of the spot_tracker library
function calls..

WARNING: All of the CUDA code for the entire project has to be in here
so that we only initialize the device once.
**********************************************************************/

#include "image_wrapper.h"
#include "spot_tracker.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

//----------------------------------------------------------------------
// Definitions and routines needed by all functions below.
//----------------------------------------------------------------------

static hipDevice_t     g_cuDevice;     // CUDA device
static hipCtx_t    g_cuContext;    // CUDA context on the device
static float        *g_cuda_fromhost_buffer = NULL;
static unsigned		g_cuda_fromhost_nx = 0;
static unsigned		g_cuda_fromhost_ny = 0;

// For the GPU code, block size and number of kernels to run to cover a whole grid.
// Initialized once in VST_ensure_cuda_ready();
static dim3         g_threads;      // 16x16x1
static dim3         g_grid;         // Computed to cover array (slightly larger than array)

// Open the CUDA device and get a context.  Also allocate a buffer of
// appropriate size.  Do this allocation only when the size of the buffer
// allocated is different from the newly-requested size.  Return false
// if we cannot get one.  This function can be called every time a
// CUDA_using function is called, but it only does the device opening
// and image-buffer allocation once.
static bool VST_ensure_cuda_ready(const VST_cuda_image_buffer &inbuf)
{
  static bool initialized = false;	// Have we initialized yet?
  static bool okay = false;			// Did the initialization work?
  if (!initialized) {
    // Whether this works or not, we'll be initialized.
    initialized = true;
    
    // Open the largest-ID CUDA device in the system
    hipInit(0);
    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    if (num_devices == 0) {
      fprintf(stderr,"VST_ensure_cuda_ready(): No CUDA devices.\n");
      return false;
    }
    if (hipDeviceGet(&g_cuDevice, num_devices-1) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get device.\n");
      return false;
    }
    if (hipCtxCreate( &g_cuContext, 0, g_cuDevice ) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get context.\n");
      return false;
    }
    
    // Allocate a buffer to be used on the GPU.  It will be
    // copied from host memory.
    if ( (inbuf.nx != g_cuda_fromhost_nx) || (inbuf.ny != g_cuda_fromhost_nx) ) {
	    
		unsigned int numBytes = inbuf.nx * inbuf.ny * sizeof(float);
		if (g_cuda_fromhost_buffer != NULL) {
			hipFree(g_cuda_fromhost_buffer);
		}
		if (hipMalloc((void**)&g_cuda_fromhost_buffer, numBytes) != hipSuccess) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Could not allocate memory.\n");
		  return false;
		}
		if (g_cuda_fromhost_buffer == NULL) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Buffer is NULL pointer.\n");
		  return false;
		}
		g_cuda_fromhost_nx = inbuf.nx;
		g_cuda_fromhost_ny = inbuf.ny;
	}
	
    // Set up enough threads (and enough blocks of threads) to at least
    // cover the size of the array.  We use a thread block size of 16x16
    // because that's what the example matrixMul code from nVidia does.
    // Changing them to 8 and 8 makes the Gaussian kernel slower.  Changing
    // them to 32 and 32 also makes it slower (by not as much)
    g_threads.x = 16;
    g_threads.y = 16;
    g_threads.z = 1;
    g_grid.x = (g_cuda_fromhost_nx / g_threads.x) + 1;
    g_grid.y = (g_cuda_fromhost_ny / g_threads.y) + 1;
    g_grid.z = 1;	

    // Everything worked, so we're okay.
    okay = true;
  }

  // Return true if we are okay.
  return okay;
}

//----------------------------------------------------------------------
// Functions called from image_wrapper.cpp.
//----------------------------------------------------------------------

// Compute the value of a Gaussian at the specified point.  The function is 2D,
// centered at the origin.  The "standard normal distribution" Gaussian has an integrated
// volume of 1 over all space and a variance of 1.  It is defined as:
//               1           -(R^2)/2
//   G(x) = ------------ * e
//             2*PI
// where R is the radius of the sample point from the origin.
// We let the user set the standard deviation s, changing the function to:
//                  1           -(R^2)/(2*s^2)
//   G(x) = --------------- * e
//           s^2 * 2*PI
// For computational efficiency, we refactor this into A * e ^ (B * R^2).

inline __device__ float	cuda_Gaussian(
  float s_meters,      //< standard deviation (square root of variance)
  float x, float y)	//< Point to sample (relative to origin)
{
  float variance = s_meters * s_meters;
  float R_squared = x*x + y*y;

  const float twoPI = static_cast<float>(2*HIP_PI_F);
  const float twoPIinv = 1.0f / twoPI;
  float B = -1 / (2 * variance);
  float A = twoPIinv / variance;

  return A * __expf(B * R_squared);
}

// CUDA kernel to do a Gaussian blur of the passed-in image and place
// it into the output images.
// Told the buffer beginning and the buffer size.  Assumes at least
// as many threads are run as there are elements in the buffer.
// Assumes a 2D array of threads.
static __global__ void VST_cuda_gaussian_blur(float *in, float *out, int nx, int ny,
							unsigned aperture, float std)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( (x < nx) && (y < ny) ) {

	// Replacing the cuda_Gaussian() calls below with 1 changed the speed from
	// 23 frames/second to 25 frames/second, so it is not the bottleneck.
	// Replacing the in[] read call with 1 slows things down to 21.
	// Replacing the inside-if with (weight++; sum++) speeds things to 29.
	// Removing the if() test speeds things up to 45 (but gets funky answers).
	// Pulling the if() statement out of the inner loop and into the bounds
	// setting for the i and j loops made the speed 39.
	// Swapping the sum += and weight += lines below to put weight later brought
	// us up to 43.
	// Swapping the kval = and value = lines to put kval first brought it up
	// to 43.8.
	// Moving the kval and value definitions outside the loop dropped back to
	// 39.
	// Moving the definition of int j into the i loop bumped it up to 44.  Looks
	// like the compiler doesn't always do the best optimizing for us...
	// After the above mods, changing the cuda_Gaussian() to = 1.0f made things
	// go 53 frames/second, so there may be some computational gain to be had
	// in there.
	// XXX Switching the code below to one like the faster algorithm in the
	// CPU code may speed things up a bit more.
	// Changing the radius (affects the aperture) from 5 to 3 makes things
	// go 41 frames/second.
    // If we don't have an integer version of aperture, the "-aperture"
    // below turns into a large positive number, meaning that
    // the loops never get executed.
    int aperture_int = aperture;
    
    // Determine the safe bounds to read from around this point.  This avoids
    // having to put an if() statement in the inner loop, which slows us down
    // a bunch.
    int min_i = -aperture_int;
    int max_i = aperture_int;
    int min_j = -aperture_int;
    int max_j = aperture_int;
    int min_x = x - aperture_int; if (min_x < 0) { min_i = -min_x; }
    int min_y = y - aperture_int; if (min_y < 0) { min_j = -min_y; }
    int max_x = x + aperture_int; if (max_x >= nx) { max_i -= ( max_x - (nx-1) ); }
    int max_y = y + aperture_int; if (max_y >= ny) { max_j -= ( max_y - (ny-1) ); }
    int i;
    float sum = 0;
    float weight = 0;
    for (i = min_i; i <= max_i; i++) {
	  int j;
      for (j = min_j; j <= max_j; j++) {
          float kval = cuda_Gaussian(std,i,j);
		  float value = in[x+i + (y+j)*nx];
          sum += kval * value;
          weight += kval;
      }
    }
	out[x + y*nx] = sum/weight;
	
  }
}

bool VST_cuda_blur_image(VST_cuda_image_buffer &buf, unsigned aperture, float std)
{
	// Make sure we can initialize CUDA.  This also allocates the global
	// input buffer that we'll copy data from the host into.
	if (!VST_ensure_cuda_ready(buf)) { return false; }
	
	// Copy the input image from host memory into the GPU buffer.
	size_t size = buf.nx * buf.ny * sizeof(float);
	if (hipMemcpy(g_cuda_fromhost_buffer, buf.buf, size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not copy memory to CUDA\n");
		return false;
	}
	
	// Allocate a CUDA buffer to blur into from the input buffer.  It should
	// be the same size as the input buffer.  We only allocate this when the
	// size changes.
	static int blur_nx = 0;
	static int blur_ny = 0;
	static float *blur_buf = NULL;
	if ( (blur_nx != g_cuda_fromhost_nx) || (blur_ny != g_cuda_fromhost_ny) ) {
		if (blur_buf != NULL) { hipFree(blur_buf); }
		blur_nx = g_cuda_fromhost_nx;
		blur_ny = g_cuda_fromhost_ny;
		if (hipMalloc((void**)&blur_buf, size) != hipSuccess) {
		  fprintf(stderr,"VST_cuda_blur_image(): Could not allocate memory.\n");
		  return false;
		}
	}
	if (blur_buf == NULL) {
	  fprintf(stderr,"VST_cuda_blur_image(): Buffer is NULL pointer.\n");
	  return false;
	}
	
	// Call the CUDA kernel to do the blurring on the image, reading from
	// the global input buffer and writing to the blur buffer.
	// Synchronize the threads when
	// we are done so we know that they finish before we copy the memory
	// back.
	VST_cuda_gaussian_blur<<< g_grid, g_threads >>>(g_cuda_fromhost_buffer,
					blur_buf, blur_nx, blur_ny, aperture, std);
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not synchronize threads\n");
		return false;
	}

	// Copy the buffer back from the GPU to host memory.
	if (hipMemcpy(buf.buf, blur_buf, size, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not copy memory back to host\n");
		return false;
	}
	
	// Everything worked!
	return true;
}

//----------------------------------------------------------------------
// Functions called from spot_tracker.cpp.
//----------------------------------------------------------------------

typedef struct {
	float radius;
	float sample_separation;
	float pixel_accuracy;
	float x;
	float y;
} CUDA_Tracker_Info;

// CUDA kernel to optimize the passed-in list of trackers based on the
// passed-in image.  Moves the X and Y position of each tracker to its
// final optimum location.
// Assumes that we have at least as many threads in X as we have trackers.
// Assumes a 2D array of threads.
static __global__ void VST_cuda_symmetric_opt_kernel(float *img, int nx, int ny,
							CUDA_Tracker_Info *tkrs, int nt)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  // For now, just do one thread per tracker and have it be the one with y=0.
  if ( (x < nt) && (y < 1) ) {
	// XXXXX

  }
}

// Optimize the passed-in list of symmetric XY trackers based on the
bool VST_cuda_optimize_symmetric_trackers(const VST_cuda_image_buffer &buf,
                                                 std::list<Spot_Information *> &tkrs,
                                                 unsigned num_to_optimize)
{
	// Make sure we can initialize CUDA.  This also allocates the global
	// input buffer that we'll copy data from the host into.
	if (!VST_ensure_cuda_ready(buf)) { return false; }
	
	// Copy the input image from host memory into the GPU buffer.
	size_t imgsize = buf.nx * buf.ny * sizeof(float);
	if (hipMemcpy(g_cuda_fromhost_buffer, buf.buf, imgsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory to CUDA\n");
		return false;
	}
	
	// Allocate an array of tracker information to pass down to the kernel.
	// with one entry per tracker we are optimizing.  This stores the tracking
	// parameters associated with each tracker along with its X and Y positions;
	// the kernel will replace the X and Y locations, which are then copied back
	// into the trackers.
	CUDA_Tracker_Info *ti = new CUDA_Tracker_Info[tkrs.size()];
	if (ti == NULL) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Out of memory\n");
		return false;
	}
	int i;
	std::list<Spot_Information *>::iterator  loop;
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		spot_tracker_XY *t = (*loop)->xytracker();
		ti[i].radius = static_cast<float>(t->get_radius());
		ti[i].sample_separation = static_cast<float>(t->get_sample_separation());
		ti[i].pixel_accuracy = static_cast<float>(t->get_pixel_accuracy());
		ti[i].x = static_cast<float>(t->get_x());
		ti[i].y = static_cast<float>(t->get_y());
	}
	
	// Allocate a GPU buffer to store the tracker information.
	// Copy the tracker information from the host to GPU memory.
	CUDA_Tracker_Info *gpu_ti;
	size_t tkrsize = num_to_optimize * sizeof(CUDA_Tracker_Info);
	if (hipMalloc((void**)&gpu_ti, tkrsize) != hipSuccess) {
	  fprintf(stderr,"VST_cuda_optimize_symmetric_trackers(): Could not allocate memory.\n");
	  delete [] ti;
	  return false;
	}
	if (hipMemcpy(gpu_ti, ti, tkrsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory to CUDA\n");
		delete [] ti;
		return false;
	}
	
    // Set up enough threads (and enough blocks of threads) to at least
    // cover the number of trackers in X, and we have only one thread per
    // tracker (so y = 1).  We use a thread block size of 1x1
    // because we don't want to share a cache between different trackers.
    g_threads.x = 1;
    g_threads.y = 1;
    g_threads.z = 1;
    g_grid.x = num_to_optimize;
    g_grid.y = 1;
    g_grid.z = 1;	
	
	// Call the CUDA kernel to do the tracking, reading from
	// the input buffer and editing the positions in place.
	// Synchronize the threads when we are done so we know that 
	// they finish before we copy the memory back.
	printf("XXX Not yet completely implemented\n");
	VST_cuda_symmetric_opt_kernel<<< g_grid, g_threads >>>(g_cuda_fromhost_buffer,
					buf.nx, buf.ny,
					gpu_ti, num_to_optimize);
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not synchronize threads\n");
		return false;
	}

	// Copy the tracker info back from the GPU to the host memory.
	if (hipMemcpy(ti, gpu_ti, tkrsize, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory back to host\n");
		hipFree(gpu_ti);
		delete [] ti;
		return false;
	}
	
	// Copy the positions back into the trackers.
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		spot_tracker_XY *t = (*loop)->xytracker();
		t->set_location(ti[i].x, ti[i].y);
	}
		
	// Free the array of tracker info on the GPU and host sides.
	hipFree(gpu_ti); gpu_ti = NULL;
	delete [] ti; ti = NULL;

	// Done!
	return true;
}

#include "hip/hip_runtime.h"
/*********************************************************************
This is the source file for the CUDA version of the spot_tracker library
function calls..

WARNING: All of the CUDA code for the entire project has to be in here
so that we only initialize the device once.
**********************************************************************/

#include "image_wrapper.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

//----------------------------------------------------------------------
// Definitions and routines needed by all functions below.
//----------------------------------------------------------------------

static hipDevice_t     g_cuDevice;     // CUDA device
static hipCtx_t    g_cuContext;    // CUDA context on the device

// Open the CUDA device and get a context.  Return false
// if we cannot get one.  This function can be called every time a
// CUDA_using function is called, but it only does the device opening
// and image-buffer allocation once.
static bool VST_ensure_cuda_ready(void)
{
  static bool initialized = false;	// Have we initialized yet?
  static bool okay = false;			// Did the initialization work?
  if (!initialized) {
    // Whether this works or not, we'll be initialized.
    initialized = true;
    
    // Open the largest-ID CUDA device in the system
    hipInit(0);
    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    if (num_devices == 0) {
      fprintf(stderr,"VST_ensure_cuda_ready(): No CUDA devices.\n");
      return false;
    }
    if (hipDeviceGet(&g_cuDevice, num_devices-1) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get device.\n");
      return false;
    }
    if (hipCtxCreate( &g_cuContext, 0, g_cuDevice ) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get context.\n");
      return false;
    }
    
    // Everything worked, so we're okay.
    okay = true;
  }

  // Return true if we are okay.
  return okay;
}

//----------------------------------------------------------------------
// Functions called from image_wrapper.cpp.
//----------------------------------------------------------------------

bool VST_cuda_blur_image(VST_cuda_image_buffer &buf, unsigned aperture, float std)
{
// XXX until we fix this to work
return false;

	// Make sure we can initialize CUDA.
	if (!VST_ensure_cuda_ready()) { return false; }
	
	// Allocate a CUDA buffer on the card to store the image.  Copy the
	// image to the buffer.
	// XXX;
	
	// Call the CUDA kernel to do the blurring on the image.
	// XXX
	
	// Copy the buffer back from the card.
	// XXX
	
	// Free the CUDA buffer that was allocated to copy to and from the
	// card.
	// XXX
	
	// Everything worked!
	return true;
}


//----------------------------------------------------------------------
// XXX Constants and stuff from Panoptes Simulator.
//----------------------------------------------------------------------

const unsigned PSCS_cols = 648;   // Image size for the camera.
const unsigned PSCS_rows = 488;
typedef float PSCS_buffer[PSCS_cols][PSCS_rows];

// Holds the information needed to render one bead.  This is a struct
// that can be used in either standard code or CUDA code, so that we
// can accelerate the simulation if needed.
typedef struct {
  float position[3];        // X, Y, Z location of the bead
  float radius;             // Radius of the bead
  float fluoro1_response;   // Number of photons emitted for a 1-second exposure, fluorophore 1
  float fluoro2_response;   // Number of photons emitted for a 1-second exposure, fluorophore 2
} PS_Bead;

static float        *g_cuda_buffer = NULL;

// For the camera simulator, block size and number of kernels to run to cover a whole grid.
// Initialized once in ensure_cuda_ready();
static dim3         g_threads;      // 16x16x1
static dim3         g_grid;         // Computed to cover array (slightly larger than array)

// Allocate a PCSC_buffer-sized element on the GPU.  Return false
// if we cannot get one.  This function can be called every time a
// CUDA_using function is called, but it only does the device opening
// and image-buffer allocation once.
static bool ensure_cuda_ready(void)
{
  static bool initialized = false;
  if (!initialized) {
    // Open the largest-ID CUDA device in the system
    hipInit(0);
    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    if (num_devices == 0) {
      fprintf(stderr,"ensure_cuda_ready(): No CUDA devices.\n");
      return false;
    }
    if (hipDeviceGet(&g_cuDevice, num_devices-1) != hipSuccess) {
      fprintf(stderr,"ensure_cuda_ready(): Could not get device.\n");
      return false;
    }
    if (hipCtxCreate( &g_cuContext, 0, g_cuDevice ) != hipSuccess) {
      fprintf(stderr,"ensure_cuda_ready(): Could not get context.\n");
      return false;
    }

    // Allocate a buffer to be used on the GPU.  It will be
    // copied back and forth from host memory.
    unsigned int numBytes = PSCS_cols * PSCS_rows * sizeof(float);
    if (hipMalloc((void**)&g_cuda_buffer, numBytes) != hipSuccess) {
      fprintf(stderr,"ensure_cuda_ready(): Could not allocate memory.\n");
      return false;
    }
    if (g_cuda_buffer == NULL) {
      fprintf(stderr,"ensure_cuda_ready(): Buffer is NULL pointer.\n");
      return false;
    }

    // Set up enough threads (and enough blocks of threads) to at least
    // cover the size of the array.  We use a thread block size of 16x16
    // because that's what the example matrixMul code from nVidia does.
    g_threads.x = 16;
    g_threads.y = 16;
    g_threads.z = 1;
    g_grid.x = (PSCS_cols / g_threads.x) + 1;
    g_grid.y = (PSCS_rows / g_threads.y) + 1;
    g_grid.z = 1;

    initialized = true;
  }

  // We're good if we have a buffer.
  return g_cuda_buffer != NULL;
}

//----------------------------------------------------------------------
// Functions for the camera simulator.
//----------------------------------------------------------------------

// CUDA kernel to clear all of the elements of the matrix to zero.
// Told the buffer beginning and the buffer size.  Assumes at least
// as many threads are run as there are elements in the buffer.
// Assumes a 2D array of threads.
static __global__ void clear_float_buffer(float *buf, int nx, int ny)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( (x < nx) && (y < ny) ) {
    buf[y + ny*x] = 0.0;
  }
}

// Clear every pixel in the image to black.  Return true on success
// and false on failure (for example, if no CUDA).
bool PSCS_cuda_clear(PSCS_buffer &buf)
{
  if (!ensure_cuda_ready()) { return false; }

  // We don't need to copy the buffer from host memory here, because
  // we're going to set it to a certain value no matter what it started
  // out as.

  // Run the CUDA kernel to clear the memory.  Uses the block and thread
  // counts found by ensure_cuda_ready().  Synchronize the threads when
  // we are done so we know that they finish before we copy the memory
  // back.
  clear_float_buffer<<< g_grid, g_threads >>>(g_cuda_buffer, PSCS_cols, PSCS_rows);
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_clear(): Could not synchronize threads\n");
    return false;
  }

  // Copy the buffer back from CUDA memory to host memory.
  size_t size = PSCS_cols * PSCS_rows * sizeof(float);
  if (hipMemcpy(buf, g_cuda_buffer, size, hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_clear(): Could not copy memory back to host\n");
    return false;
  }

  // Done!
  return true;
}

// CUDA kernel to add to all of the elements of the matrix.
// Told the buffer beginning and the buffer size.  Assumes at least
// as many threads are run as there are elements in the buffer.
// Assumes a 2D array of threads.
static __global__ void add_to_float_buffer(float *buf, int nx, int ny, float increment)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( (x < nx) && (y < ny) ) {
    buf[y + ny*x] += increment;
  }
}

// Add the specified amount of photons to the current value of the buffer that
// is passed in.  Return true on success
// and false on failure (for example, if no CUDA).
bool PSCS_cuda_accumulate_brightfield_photons(PSCS_buffer &buf, float photons)
{
  if (!ensure_cuda_ready()) { return false; }

  // Copy the buffer from host memory to CUDA memory so we have something to
  // add to.
  size_t size = PSCS_cols * PSCS_rows * sizeof(float);
  if (hipMemcpy(g_cuda_buffer, buf, size, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_accumulate_brightfield_photons(): Could not copy memory to CUDA\n");
    return false;
  }

  // Run the CUDA kernel to add to the memory.  Uses the block and thread
  // counts found by ensure_cuda_ready().  Synchronize the threads when
  // we are done so we know that they finish before we copy the memory
  // back.
  add_to_float_buffer<<< g_grid, g_threads >>>(g_cuda_buffer, PSCS_cols, PSCS_rows, photons);
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_accumulate_brightfield_photons(): Could not synchronize threads\n");
    return false;
  }

  // Copy the buffer back from CUDA memory to host memory.
  if (hipMemcpy(buf, g_cuda_buffer, size, hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_accumulate_brightfield_photons(): Could not copy memory back to host\n");
    return false;
  }

  // Done!
  return true;
}

// Compute the value of a Gaussian at the specified point.  The function is 2D,
// centered at the origin.  The "standard normal distribution" Gaussian has an integrated
// volume of 1 over all space and a variance of 1.  It is defined as:
//               1           -(R^2)/2
//   G(x) = ------------ * e
//             2*PI
// where R is the radius of the sample point from the origin.
// We let the user set the standard deviation s, changing the function to:
//                  1           -(R^2)/(2*s^2)
//   G(x) = --------------- * e
//           s^2 * 2*PI
// For computational efficiency, we refactor this into A * e ^ (B * R^2).

inline __device__ float	cuda_Gaussian(
  float s_meters,      //< standard deviation (square root of variance)
  float x, float y)	//< Point to sample (relative to origin)
{
  float variance = s_meters * s_meters;
  float R_squared = x*x + y*y;

  const float twoPI = static_cast<float>(2*HIP_PI_F);
  const float twoPIinv = 1.0f / twoPI;
  float A = twoPIinv / variance;
  float B = -1 / (2 * variance);

  return A * __expf(B * R_squared);
}

// CUDA kernel to add sums of Gaussians to the image.
// Told the buffer beginning and the buffer size.
// The count of how many Gaussians there are is followed by a
// floating-point 1D array that has four entries in it for each bead: the X and Y
// position, the radius, and the value of the Gaussian.  Assumes at least
// as many threads are run as there are elements in the buffer.
// Assumes a 2D array of threads.
static __global__ void add_gaussians_to_float_buffer(float *buf, int nx, int ny,
                                                     size_t count, float *params)
{
  // Find out which pixel I'm responsible for.
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // For each Gaussian in the list, add its contribution to my pixel
  // based on the distance from my pixel and it.  Only do this if my
  // pixel is within the image.
  if ( (x < nx) && (y < ny) ) {
    size_t i;
    for (i = 0; i < count; i++) {
      float gx = params[4*i + 0];
      float gy = params[4*i + 1];
      float r = params[4*i + 2];
      float v = params[4*i + 3];
      buf[y + ny*x] += v * cuda_Gaussian(r, gx-x, gy-y);
    }
  }
}

// Sum the set of Gaussians whose parameters are passed in into the floating-point
// buffer passed in.  The count of how many Gaussians there are is followed by a
// floating-point 1D array that has four entries in it for each bead: the X and Y
// position, the radius, and the value of the Gaussian.  Return true on success
// and false on failure (for example, if no CUDA).
bool PSCS_cuda_accumulate_gaussians(PSCS_buffer &buf, size_t count, float *params)
{
  if (!ensure_cuda_ready()) { return false; }
  //return false;

  // We need to copy the bead-parameter buffer from host memory to CUDA
  // memory before calling the kernel.  But we need to make sure there is
  // enough room for the buffer.  To avoid having to re-allocate a buffer
  // each time we are called, we keep a static buffer around and make it
  // bigger whenever we have to.
  static size_t max_bead_params = 0;
  static float  *cuda_bead_params = NULL;
  size_t psize = 4 * count * sizeof(float);
  if (count > max_bead_params) {
    max_bead_params = count;
    if (cuda_bead_params != NULL) {
      hipFree(cuda_bead_params);
    }
    if (hipMalloc((void**)&cuda_bead_params, psize) != hipSuccess) {
      fprintf(stderr, "PSCS_cuda_accumulate_gaussians(): Out of CUDA memory\n");
      return false;
    }
  }
  if (hipMemcpy(cuda_bead_params, params, psize, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_accumulate_gaussians(): Could not copy parameters to CUDA\n");
    return false;
  }

  // Copy the buffer from host memory to CUDA memory so we have something to
  // add to.
  size_t size = PSCS_cols * PSCS_rows * sizeof(float);
  if (hipMemcpy(g_cuda_buffer, buf, size, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_accumulate_gaussians(): Could not copy memory to CUDA\n");
    return false;
  }

  // Run the CUDA kernel to add the Gaussians to the memory.  Uses the block and thread
  // counts found by ensure_cuda_ready().  Synchronize the threads when
  // we are done so we know that they finish before we copy the memory
  // back.
  add_gaussians_to_float_buffer<<< g_grid, g_threads >>>(g_cuda_buffer, PSCS_cols, PSCS_rows, count, cuda_bead_params);
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_accumulate_gaussians(): Could not synchronize threads\n");
    return false;
  }

  // Copy the buffer back from CUDA memory to host memory.
  if (hipMemcpy(buf, g_cuda_buffer, size, hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "PSCS_cuda_accumulate_gaussians(): Could not copy memory back to host\n");
    return false;
  }

  return true;
}

//----------------------------------------------------------------------
// Functions for the bead simulator.
//----------------------------------------------------------------------

//----------------------------------------------------------------------
// Helper functions

// Horrible random-number generator.  May work to some extent if you do modulo
// some reasonable number.  Does make things jump around.
// When I tried more-sophisticated ones from the web, they locked up Windows.
inline __device__ unsigned quickrand(unsigned &seed, const unsigned a, const unsigned c)
{
    seed = seed * a + c;
    return seed;
}

// Polar method for normal density discussed in Knuth.
// This one is passed two random numbers between 0 and 1
// and it turns them into a random normally-distributed
// number.

inline __device__ float random_normal_sample(unsigned &seed, unsigned a, unsigned c)
{
  float u1, u2, v1, v2;
  float S = 2;
  while (S >= 1) {
    u1 = (quickrand(seed, a, c) % 10000) / 9999.0f;
    u2 = (quickrand(seed, a, c) % 10000) / 9999.0f;
    v1 = 2*u1 - 1;
    v2 = 2*u2 - 1;
    S = v1*v1 + v2*v2;
  };
  return v1*sqrtf( (-2*__logf(S))/S );
}

// CUDA kernel to add random offsets to a list of beads
// Told the buffer beginning and the buffer size.
// Assumes a 1D array of threads.  Assumes as many random
// number seeds as there are blocks.
static __global__ void move_beads_kernel(PS_Bead *buf, size_t count, float delta_t,
                                         float mean_r,
                                         float motion_x, float motion_y, float motion_z,
                                         unsigned *seeds)
{
  // Initializing the random-number generator using the
  // seeds we got from the host.  We seed based on our
  // block ID and then run it forward based on our thread
  // ID.
  unsigned a = 4529;
  unsigned c = 19723;
  unsigned seed = seeds[blockIdx.x];

  // Find out which bead I'm responsible for.
  size_t me = blockIdx.x * blockDim.x + threadIdx.x;

  // Randomly move the bead I am responsible based on the time interval
  // and its characteristics.
  if (me < count) {
    // Compute the change in coefficient to be taken based on the
    // difference between one second (the duration for which the
    // motion parameter was specified) and the actual time period
    // asked for.

    // The root mean squared displacement of a bead over time is
    // given by sqrt( 2 * D * t ), where t is the duration in time
    // between measurements.  Therefore, the time step should be
    // divided by the square-root of 1 (which is 1) and multiplied
    // by the square-root of delta_t.

    float time_scale = sqrtf(delta_t);

    // Compute change in coefficient to be made based on the
    // difference between the mean and actual parameter for this
    // bead.  In the above equation, D is the diffusion coefficient
    // and D = kB * T / b, where only b depends on the bead radius,
    // and b = 6 * pi * viscosity * r, where r is the bead radius.
    // Keeping all other terms constant, this means that the change
    // due to r in D is an inverse relationship, as is the change
    // with respect to D.  Do the final change is the square root
    // of the inverse change: delta = sqrt( (1/r) / (1/mean) )
    // = sqrt( mean/r )

    float radius_scale = sqrtf( mean_r / buf[me].radius );

    // Scale the standard deviation motion parameter by the above
    // scales and then sample from a Gaussian with that standard
    // deviation for each axis.

    float std_x = motion_x * time_scale * radius_scale;
    float std_y = motion_y * time_scale * radius_scale;
    float std_z = motion_z * time_scale * radius_scale;

    // Get three random normal samples.  We have to pass a random
    // number between 0 and 1 to each, which we get by scaling the
    // unsigned random numbers.
    float normal_samples[3] = {0, 0, 0};
    unsigned i;
    for (i = 0; i < 3; i++) {
      normal_samples[i] = random_normal_sample(seed, a, c);
    }
    buf[me].position[0] += std_x * normal_samples[0];
    buf[me].position[1] += std_y * normal_samples[1];
    buf[me].position[2] += std_z * normal_samples[2];
  }
}

// Move the beads whose parameters are passed in into the floating-point
// buffer passed in.  Return true on success
// and false on failure (for example, if no CUDA).
bool PSBS_cuda_move_beads(PS_Bead *buf, unsigned count, float delta_t,
                          float mean_r, float motion[3])
{
  if (!ensure_cuda_ready()) { return false; }

  unsigned threads = 256;
  unsigned blocks = (count / threads) + 1; // Make sure we have at least enough.

  // Allocate a block of random-number seeds on CUDA, which we initialize
  // each time we are called.  We need to have as many seeds as there are
  // blocks.  We re-allocate if we don't have enough.
  static unsigned *cuda_seeds = NULL;
  static size_t max_seeds = 0;

  size_t rsize = blocks * sizeof(unsigned);
  if (blocks > max_seeds) {
    max_seeds = blocks;
    if (cuda_seeds != NULL) {
      hipFree(cuda_seeds);
    }
    if (hipMalloc((void**)&cuda_seeds, rsize) != hipSuccess) {
      fprintf(stderr, "PSBS_cuda_move_beads(): Out of CUDA memory\n");
      return false;
    }
  }

  // Generate a list of random-number seeds, one per thread.  Pass this list down to
  // CUDA memory.
  unsigned *seeds = new unsigned[blocks];
  if (seeds == NULL) {
    fprintf(stderr, "PSBS_cuda_move_beads(): Out of memory\n");
    return false;
  }
  unsigned i;
  for (i = 0; i < blocks; i++) {
    seeds[i] = rand();
  }
  if (hipMemcpy(cuda_seeds, seeds, rsize, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "PSBS_cuda_move_beads(): Could not copy random seeds to CUDA\n");
    return false;
  }
  delete [] seeds;

  // We need to copy the bead-parameter buffer from host memory to CUDA
  // memory before calling the kernel.  But we need to make sure there is
  // enough room for the buffer.  To avoid having to re-allocate a buffer
  // each time we are called, we keep a static buffer around and make it
  // bigger whenever we have to.
  static size_t max_bead_params = 0;
  static PS_Bead  *cuda_bead_params = NULL;
  size_t psize = count * sizeof(PS_Bead);
  if (count > max_bead_params) {
    max_bead_params = count;
    if (cuda_bead_params != NULL) {
      hipFree(cuda_bead_params);
    }
    if (hipMalloc((void**)&cuda_bead_params, psize) != hipSuccess) {
      fprintf(stderr, "PSBS_cuda_move_beads(): Out of CUDA memory\n");
      return false;
    }
  }
  if (hipMemcpy(cuda_bead_params, buf, psize, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "PSBS_cuda_move_beads(): Could not copy parameters to CUDA\n");
    return false;
  }

  // Run the CUDA kernel to move the beads.  Synchronize the threads when
  // we are done so we know that they finish before we copy the memory
  // back.
  move_beads_kernel<<< blocks, threads >>>(cuda_bead_params, count, delta_t,
    mean_r, motion[0], motion[1], motion[2], cuda_seeds);
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "PSBS_cuda_move_beads(): Could not synchronize threads\n");
    return false;
  }

  // Copy the buffer back from CUDA memory to host memory.
  if (hipMemcpy(buf, cuda_bead_params, psize, hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "PSBS_cuda_move_beads(): Could not copy parameters back to host\n");
    return false;
  }

  return true;
}

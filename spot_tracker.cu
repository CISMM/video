#include "hip/hip_runtime.h"
/*********************************************************************
This is the source file for the CUDA version of the spot_tracker library
function calls..

WARNING: All of the CUDA code for the entire project has to be in here
so that we only initialize the device once.
**********************************************************************/

#include "image_wrapper.h"
#include "spot_tracker.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

//----------------------------------------------------------------------
// Definitions and routines needed by all functions below.
//----------------------------------------------------------------------

const int MAX_LATTICE = 8;	// Maximum lattice size for tracker optimization

static hipDevice_t     g_cuDevice;     // CUDA device
static hipCtx_t    g_cuContext;    // CUDA context on the device

#define USE_TEXTURE
#ifdef USE_TEXTURE
static hipArray	*g_cuda_fromhost_array = NULL;
static hipChannelFormatDesc	g_channel_desc;
static texture<float,hipTextureType2D,hipReadModeElementType>	g_tex_ref;
#else
static float        *g_cuda_fromhost_buffer = NULL;
#endif

static unsigned		g_cuda_fromhost_nx = 0;
static unsigned		g_cuda_fromhost_ny = 0;

// For the GPU code, block size and number of kernels to run to cover a whole grid.
// Initialized once in VST_ensure_cuda_ready();
static dim3         g_threads;      // 16x16x1
static dim3         g_grid;         // Computed to cover array (slightly larger than array)

// Open the CUDA device and get a context.  Also allocate a buffer of
// appropriate size.  Do this allocation only when the size of the buffer
// allocated is different from the newly-requested size.  Return false
// if we cannot get one.  This function can be called every time a
// CUDA_using function is called, but it only does the device opening
// and image-buffer allocation once.
static bool VST_ensure_cuda_ready(const VST_cuda_image_buffer &inbuf)
{
  static bool initialized = false;	// Have we initialized yet?
  static bool okay = false;		// Did the initialization work?
  if (!initialized) {
    // Whether this works or not, we'll be initialized.
    initialized = true;
    
    // Open the largest-ID CUDA device in the system
    hipInit(0);
    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    if (num_devices == 0) {
      fprintf(stderr,"VST_ensure_cuda_ready(): No CUDA devices.\n");
      return false;
    }
    if (hipDeviceGet(&g_cuDevice, num_devices-1) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get device.\n");
      return false;
    }
    if (hipCtxCreate( &g_cuContext, 0, g_cuDevice ) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get context.\n");
      return false;
    }
    
    // Allocate a buffer to be used on the GPU.  It will be
    // copied from host memory.
    if ( (inbuf.nx != g_cuda_fromhost_nx) || (inbuf.ny != g_cuda_fromhost_ny) ) {
	    
#ifdef	USE_TEXTURE
		if (g_cuda_fromhost_array != NULL) {
			hipFreeArray(g_cuda_fromhost_array);
		}
		// 32-bit floating-point values in the first texture component only.
		g_channel_desc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
		if (hipMallocArray(&g_cuda_fromhost_array, &g_channel_desc, inbuf.nx, inbuf.ny)  != hipSuccess) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Could not allocate array.\n");
		  return false;
		}
		if (g_cuda_fromhost_array == NULL) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Array is NULL pointer.\n");
		  return false;
		}
#else
		unsigned int numBytes = inbuf.nx * inbuf.ny * sizeof(float);
		if (g_cuda_fromhost_buffer != NULL) {
			hipFree(g_cuda_fromhost_buffer);
		}
		if (hipMalloc((void**)&g_cuda_fromhost_buffer, numBytes) != hipSuccess) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Could not allocate memory.\n");
		  return false;
		}
		if (g_cuda_fromhost_buffer == NULL) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Buffer is NULL pointer.\n");
		  return false;
		}
#endif
		g_cuda_fromhost_nx = inbuf.nx;
		g_cuda_fromhost_ny = inbuf.ny;
	}
	
    // Everything worked, so we're okay.
    okay = true;
  }

  // Return true if we are okay.
  return okay;
}

//----------------------------------------------------------------------
// Functions called from image_wrapper.cpp.
//----------------------------------------------------------------------

// Compute the value of a Gaussian at the specified point.  The function is 2D,
// centered at the origin.  The "standard normal distribution" Gaussian has an integrated
// volume of 1 over all space and a variance of 1.  It is defined as:
//               1           -(R^2)/2
//   G(x) = ------------ * e
//             2*PI
// where R is the radius of the sample point from the origin.
// We let the user set the standard deviation s, changing the function to:
//                  1           -(R^2)/(2*s^2)
//   G(x) = --------------- * e
//           s^2 * 2*PI
// For computational efficiency, we refactor this into A * e ^ (B * R^2).

inline __device__ float	cuda_Gaussian(
  const float &s_meters,      //< standard deviation (square root of variance)
  const float &x, const float &y)	//< Point to sample (relative to origin)
{
  float variance = s_meters * s_meters;
  float R_squared = x*x + y*y;

  const float twoPI = static_cast<float>(2*HIP_PI_F);
  const float twoPIinv = 1.0f / twoPI;
  float B = -1 / (2 * variance);
  float A = twoPIinv / variance;

  return A * __expf(B * R_squared);
}

// CUDA kernel to do a Gaussian blur of the passed-in image and place
// it into the output images.
// Told the buffer beginning and the buffer size.  Assumes at least
// as many threads are run as there are elements in the buffer.
// Assumes a 2D array of threads.
#ifdef	USE_TEXTURE
static __global__ void VST_cuda_gaussian_blur(float *out, int nx, int ny,
							unsigned aperture, float std)
#else
static __global__ void VST_cuda_gaussian_blur(const float *in, float *out, int nx, int ny,
							unsigned aperture, float std)
#endif
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if ( (x < nx) && (y < ny) ) {

    // Replacing the cuda_Gaussian() calls below with 1 changed the speed from
    // 23 frames/second to 25 frames/second, so it is not the bottleneck.
    // Replacing the in[] read call with 1 slows things down to 21.
    // Replacing the inside-if with (weight++; sum++) speeds things to 29.
    // Removing the if() test speeds things up to 45 (but gets funky answers).
    // Pulling the if() statement out of the inner loop and into the bounds
    // setting for the i and j loops made the speed 39.
    // Swapping the sum += and weight += lines below to put weight later brought
    // us up to 43.
    // Swapping the kval = and value = lines to put kval first brought it up
    // to 43.8.
    // Moving the kval and value definitions outside the loop dropped back to
    // 39.
    // Changing the radius (affects the aperture) from 5 to 3 makes things
    // go 59 frames/second.
    // Moving the definition of int j into the i loop bumped it up to 44.  Looks
    // like the compiler doesn't always do the best optimizing for us...
    // After the above mods, changing the cuda_Gaussian() to = 1.0f made things
    // go 53 frames/second, so there may be some computational gain to be had
    // in there.
    // Switching to texture reads kept us at 44 frames/second (same 44.5 as floats).
    // XXX Switching the code below to one like the faster algorithm in the
    // CPU code may speed things up a bit more.
    // If we don't have an integer version of aperture, the "-aperture"
    // below turns into a large positive number, meaning that
    // the loops never get executed.
    int aperture_int = aperture;
    
    // Determine the safe bounds to read from around this point.  This avoids
    // having to put an if() statement in the inner loop, which slows us down
    // a bunch.
    int min_i = -aperture_int;
    int max_i = aperture_int;
    int min_j = -aperture_int;
    int max_j = aperture_int;
    int min_x = x - aperture_int; if (min_x < 0) { min_i += -min_x; }
    int min_y = y - aperture_int; if (min_y < 0) { min_j += -min_y; }
    int max_x = x + aperture_int; if (max_x >= nx) { max_i -= ( max_x - (nx-1) ); }
    int max_y = y + aperture_int; if (max_y >= ny) { max_j -= ( max_y - (ny-1) ); }
    int i;
    float sum = 0;
    float weight = 0;
    for (i = min_i; i <= max_i; i++) {
      int j;
      for (j = min_j; j <= max_j; j++) {
          float kval = cuda_Gaussian(std,i,j);
#ifdef	USE_TEXTURE
	  // Because array indices are not normalized to [0,1], we need to
	  // add 0.5f to each coordinate (quirk inherited from graphics).
	  float value = tex2D( g_tex_ref, x+i+0.5f, y+j+0.5f  );
#else
	  float value = in[x+i + (y+j)*nx];
#endif
          sum += kval * value;
          weight += kval;
      }
    }
    out[x + y*nx] = sum/weight;
  }
}

bool VST_cuda_blur_image(VST_cuda_image_buffer &buf, unsigned aperture, float std)
{
	// Make sure we can initialize CUDA.  This also allocates the global
	// input buffer that we'll copy data from the host into.
	if (!VST_ensure_cuda_ready(buf)) { return false; }

	// Copy the input image from host memory into the GPU buffer.
	size_t size = buf.nx * buf.ny * sizeof(float);
#ifdef USE_TEXTURE
	if (hipMemcpyToArray(g_cuda_fromhost_array, 0, 0, buf.buf, size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not copy array to CUDA\n");
		return false;
	}

	// Bind the texture reference to the texture after describing it.
	g_tex_ref.addressMode[0] = hipAddressModeClamp;
	g_tex_ref.addressMode[1] = hipAddressModeClamp;
	g_tex_ref.filterMode = hipFilterModePoint;
	g_tex_ref.normalized = false;
	hipBindTextureToArray(g_tex_ref, g_cuda_fromhost_array, g_channel_desc);
#else
	if (hipMemcpy(g_cuda_fromhost_buffer, buf.buf, size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not copy memory to CUDA\n");
		return false;
	}
#endif

	// Allocate a CUDA buffer to blur into from the input buffer.  It should
	// be the same size as the input buffer.  We only allocate this when the
	// size changes.
	static int blur_nx = 0;
	static int blur_ny = 0;
	static float *blur_buf = NULL;
	if ( (blur_nx != g_cuda_fromhost_nx) || (blur_ny != g_cuda_fromhost_ny) ) {
		if (blur_buf != NULL) { hipFree(blur_buf); }
		blur_nx = g_cuda_fromhost_nx;
		blur_ny = g_cuda_fromhost_ny;
		if (hipMalloc((void**)&blur_buf, size) != hipSuccess) {
		  fprintf(stderr,"VST_cuda_blur_image(): Could not allocate memory.\n");
		  return false;
		}
	}
	if (blur_buf == NULL) {
	  fprintf(stderr,"VST_cuda_blur_image(): Buffer is NULL pointer.\n");
	  return false;
	}

	// Set up enough threads (and enough blocks of threads) to at least
	// cover the size of the array.  We use a thread block size of 16x16
	// because that's what the example matrixMul code from nVidia does.
	// Changing them to 8 and 8 makes the Gaussian kernel slower.  Changing
	// them to 32 and 32 also makes blurring slower (by not as much)
	g_threads.x = 16;
	g_threads.y = 16;
	g_threads.z = 1;
	g_grid.x = ((g_cuda_fromhost_nx-1) / g_threads.x) + 1;
	g_grid.y = ((g_cuda_fromhost_ny-1) / g_threads.y) + 1;
	g_grid.z = 1;	

	// Call the CUDA kernel to do the blurring on the image, reading from
	// the global input buffer and writing to the blur buffer.
	// Synchronize the threads when
	// we are done so we know that they finish before we copy the memory
	// back.
#ifdef USE_TEXTURE
	VST_cuda_gaussian_blur<<< g_grid, g_threads >>>(blur_buf, blur_nx, blur_ny, aperture, std);
#else
	VST_cuda_gaussian_blur<<< g_grid, g_threads >>>(g_cuda_fromhost_buffer,
					blur_buf, blur_nx, blur_ny, aperture, std);
#endif
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not synchronize threads\n");
		return false;
	}

	// Copy the buffer back from the GPU to host memory.
	if (hipMemcpy(buf.buf, blur_buf, size, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not copy memory back to host\n");
		return false;
	}
	
	// Everything worked!
	return true;
}

//----------------------------------------------------------------------
// Functions called from spot_tracker.cpp.
//----------------------------------------------------------------------

typedef struct {
	float radius;
	float sample_separation;
	float pixel_accuracy;
	float x;
	float y;
	float fitness;
	bool  lost;
} CUDA_Tracker_Info;

// Find the maximum of three elements.  Return the
// index of which one was picked.
inline __device__ float max3(const float &v0, const float &v1, const float &v2,
							 unsigned &index)
{
  float max = v0; index = 0;
  if (v1 > max) { max = v1; index = 1; }
  if (v2 > max) { max = v2; index = 2; }
  return max;
}

// Read a pixel value from the specified image.
// Return false if the coordinate its outside the image.
// Return the correct result and true if the coordinate is inside.
inline __device__ bool cuda_read_pixel(
	const float *img, const int &nx, const int &ny,
	const int &x, const int &y,
	float &result)
{
	if ( (x < 0) || (y < 0) || (x >= nx) || (y >= ny) ) {
		return false;
	}
	result = img[x + y*nx];
	return true;
}

// Do bilinear interpolation to read from the image, in order to
// smoothly interpolate between pixel values.
// All sorts of speed tweaks in here because it is in the inner loop for
// the spot tracker and other codes.
// Return a result of zero and false if the coordinate its outside the image.
// Return the correct interpolated result and true if the coordinate is inside.
// Use the hardware-accelerated bilerp function by using
// a texture to read from; beware that the interpolator will only do a maximum
// of 512 subpixel locations, which won't get us below a resolution of 1/256.

inline __device__ bool cuda_read_pixel_bilerp(
	const float *img, const int &nx, const int &ny,
	const float &x, const float &y,
	float &result)
{
	result = 0;	// In case of failure.
	// The order of the following statements is optimized for speed.
	// The float version is used below for xlowfrac comp, ixlow also used later.
	// Slightly faster to explicitly compute both here to keep the answer around.
	float xlow = floor(x); int ixlow = (int)xlow;
	// The float version is used below for ylowfrac comp, ixlow also used later
	// Slightly faster to explicitly compute both here to keep the answer around.
	float ylow = floor(y); int iylow = (int)ylow;
	int ixhigh = ixlow+1;
	int iyhigh = iylow+1;
	float xhighfrac = x - xlow;
	float yhighfrac = y - ylow;
	float xlowfrac = 1.0 - xhighfrac;
	float ylowfrac = 1.0 - yhighfrac;
	float ll, lh, hl, hh;

	// Combining the if statements into one using || makes it slightly slower.
	// Interleaving the result calculation with the returns makes it slower.
	if (!cuda_read_pixel(img, nx, ny, ixlow, iylow, ll)) { return false; }
	if (!cuda_read_pixel(img, nx, ny, ixlow, iyhigh, lh)) { return false; }
	if (!cuda_read_pixel(img, nx, ny, ixhigh, iylow, hl)) { return false; }
	if (!cuda_read_pixel(img, nx, ny, ixhigh, iyhigh, hh)) { return false; }
	result = ll * xlowfrac * ylowfrac + 
		 lh * xlowfrac * yhighfrac +
		 hl * xhighfrac * ylowfrac +
		 hh * xhighfrac * yhighfrac;
	return true;
};

// Check the fitness of the specified symmetric tracker within the
// specified image at the specified location.
// This code should compute exactly the same thing as the
// symmetric_spot_tracker_interp function.
// XXX Later, put the locations to search into constant memory rather
// than computing them on the fly here.
#ifdef	USE_TEXTURE
inline __device__ float	cuda_check_fitness_symmetric(
	const hipArray *img, const int &nx, const int &ny,
	const CUDA_Tracker_Info &tkr)
#else
inline __device__ float	cuda_check_fitness_symmetric(
	const float *img, const int &nx, const int &ny,
	const CUDA_Tracker_Info &tkr)
#endif
{
	// Construct aliases for the parameters that give us easy local names.
	const float &radius = tkr.radius;
	const float &samplesep = tkr.sample_separation;
	const float &x = tkr.x;
	const float &y = tkr.y;
	
	// Sum up over rings that are samplesep away from the center; we
	// don't count the center pixel because it will never have variance.
	float r;
	float ring_variance_sum = 0;
	for (r = samplesep; r <= radius; r += samplesep) {
		float count = 0.000001f;	// Avoids need for divide-by-zero check
		float valSum = 0.0f;
		float squareValSum = 0.0f;
		float rads_per_step = samplesep / r;
		float start = (r/samplesep)*rads_per_step*0.5f;
		float theta;
		// We use the fact that sin(theta+PI) = -sin(theta) and
		// cos(theta+PI) = -cos(theta) to only have to calculate the
		// sin and cosine half as many times -- we go halfway around the
		// circle and use two points per step on opposite sides.
		for (theta = start; theta < HIP_PI_F + start; theta += rads_per_step) {
			float sintheta, costheta;
			sincos(theta, &sintheta, &costheta);

			// Do the point on the top half of the circle.
			float newx = x + r*costheta;
			float newy = y + r*sintheta;
			float val;
#ifdef	USE_TEXTURE
			// Because array indices are not normalized to [0,1], we need to
			// add 0.5f to each coordinate (quirk inherited from graphics).
			// This is doing bilinear interpolation because the g_tex_ref
			// we're using was set up to do this before our kernel was called.
			// Texture read will clamp if out of bounds.
			// This order of operations is faster than only setting val
			// once we know that we are in bounds.
			val = tex2D( g_tex_ref, newx+0.5f, newy+0.5f );
			if ( (newx >= 0) && (newy >= 0) && (newx < nx) && (newy < ny) ) {
#else
			if (cuda_read_pixel_bilerp(img, nx, ny, newx, newy, val)) {
#endif
				// Reordering these three lines makes no speed difference.
				count++;
				valSum += val;
				squareValSum += val*val;
			}

			// Do the point on the bottom half of the circle.
			newx = x - r*costheta;
			newy = y - r*sintheta;
#ifdef	USE_TEXTURE
			// Because array indices are not normalized to [0,1], we need to
			// add 0.5f to each coordinate (quirk inherited from graphics).
			// This is doing bilinear interpolation because the g_tex_ref
			// we're using was set up to do this before our kernel was called.
			// Texture read will clamp if out of bounds.
			// This order of operations is faster than only setting val
			// once we know that we are in bounds.
			val = tex2D( g_tex_ref, newx+0.5f, newy+0.5f );
			if ( (newx >= 0) && (newy >= 0) && (newx < nx) && (newy < ny) ) {
#else
			if (cuda_read_pixel_bilerp(img, nx, ny, newx, newy, val)) {
#endif
				// Reordering these three lines makes no speed difference.
				count++;
				valSum += val;
				squareValSum += val*val;
			}
		}
	
		ring_variance_sum += squareValSum - valSum*valSum / count;
	}
	
	return -ring_variance_sum;
}

// CUDA kernel to optimize the passed-in list of trackers based on the
// passed-in image.  Moves the X and Y position of each tracker to its
// final optimum location.
// Assumes a 2D array of threads.
// Assumes that we have EXACTLY as many blocks in X as we have trackers.
// Assumes a lattice of threads in X and Y that is square.
#ifdef	USE_TEXTURE
static __global__ void VST_cuda_symmetric_opt_kernel(const hipArray *img, int nx, int ny,
							CUDA_Tracker_Info *tkrs, int nt)
#else
static __global__ void VST_cuda_symmetric_opt_kernel(const float *img, int nx, int ny,
							CUDA_Tracker_Info *tkrs, int nt)
#endif
{
  // All of the threads within one block access the same tracker.
  // There is only one block in Y; there are as many blocks in X as there
  // are trackers.  We ensure that the block is square and we sample on
  // a lattice with that many points on it.
  int tkr_id = blockIdx.x;
  int my_x = threadIdx.x;
  int my_y = threadIdx.y;
  int lattice = blockDim.x;
  if (blockDim.y != blockDim.x) { return; }
  if (lattice > MAX_LATTICE) { return; }

  // This is shared memory among the threads in a block that stores the
  // position offset for each group member and the fitness that was
  // found by that group member at its position.  We fill in the offsets
  // here, each member doing its own; they make a lattice from (-,-) to
  // (+,+) where the corners are normalized from -1 to 1, and are later
  // scaled by the current step size so that the last one is at the
  // step location.  Need to subtract 1.0f rather than 1 from lattice to
  // make sure we're using floating-point division.
  // This storage must be as large as the lattice.
  __shared__ float	dx[MAX_LATTICE][MAX_LATTICE], dy[MAX_LATTICE][MAX_LATTICE];
  __shared__ float	fitnesses[MAX_LATTICE][MAX_LATTICE];
  __shared__ float	pixelstep;
  __shared__ bool	done;

  dx[my_x][my_y] = (2.0f * my_x/(lattice - 1.0f) - 1.0f);
  dy[my_x][my_y] = (2.0f * my_y/(lattice - 1.0f) - 1.0f);

  // Synchronize all of the threads in the block.
  __syncthreads();

  // Do the whole optimization here, checking with smaller and smaller
  // steps until we reach the accuracy requested.
	// Get local aliases for the passed-in variables we need to use.    
	CUDA_Tracker_Info &t = tkrs[tkr_id];
	const float &accuracy = t.pixel_accuracy;

	// Find out the initial fitness value at the present location
	// for the tracker and set done to false if I'm the first thread
	// in the block.  Also set the pixel step size to its initial
	// value of 2.0.
	if ( (my_x == 0) && (my_y == 0) ) {
		pixelstep = 2.0f;	// Start with a large value.
		t.fitness = cuda_check_fitness_symmetric(img, nx, ny, t);
		done = false;
	}

	// Synchronize all of the threads in the block.
	__syncthreads();

	// Make my own tracker with its information copied from the original
	// passed-in tracker I'm associated with.
	CUDA_Tracker_Info member_t = t;

	// Synchronize all of the threads in the block.
	__syncthreads();

	// Compute the fitness for all offsets and then figure out which
	// is the best.  If the best location is not found on the outside
	// of the lattice, then we go ahead and divide by the lattice size
	// before doing the next step.
	do {	
		// Check all offsets and compute their fitness.
		member_t.x = t.x + dx[my_x][my_y] * pixelstep;
		member_t.y = t.y + dy[my_x][my_y] * pixelstep;
		fitnesses[my_x][my_y] = cuda_check_fitness_symmetric(img, nx, ny, member_t);
		
		// Synchronize all of the threads in the block.
		__syncthreads();

		// In the first thread, find the highest fitness and its index.
		// Compare this with the original fitness.  If one is better, set
		// its value as our next start and try again.  If none are better,
		// divide the pixelstep and try again until we have a step that is
		// below our requested accuracy.  If we found a better one at the
		// edge, then we repeat our test with the same step size.
		// XXX As we have larger numbers of threads, maybe do a faster
		// parallel reduction on these values.
		if ( (my_x == 0) && (my_y == 0) ) {
			int x,y;
			int best_x = -1;
			int best_y = -1;
			float best_fitness = t.fitness;
			for (x = 0; x < lattice; x++) {
			  for (y = 0; y < lattice; y++) {
				if (fitnesses[x][y] > best_fitness) {
					best_x = x;
					best_y = y;
					best_fitness = fitnesses[x][y];
				}
			  }
			}
			// If we found a better place, move there.
			if (best_x >= 0) {
				t.x += dx[best_x][best_y] * pixelstep;
				t.y += dy[best_x][best_y] * pixelstep;
				t.fitness = best_fitness;
			}

			// If we didn't find a better place at the edge of the
			// lattice (including not finding one at all), then go ahead
			// and divide the pixel step by the lattice size, so we
			// confine our search around the best point on this lattice
			// the next time around.
			if ( (best_x != 0) && (best_x != (lattice-1)) &&
				 (best_y != 0) && (best_y != (lattice-1)) ) {

				// If our just-checked size (including the lattice) is
				// below the required accuracy, we're done.
				// XXX We need to be careful here -- the interval size
				// is not always the lattice -- it is for 2 but not for
				// higher ones (there are N-1 gaps, not N).  This code will
				// fail for a 2x2 lattice, but works for the others.
				if ( (pixelstep / (lattice-1)) <= (accuracy/2.0f) ) {
					done = true;
				}
				
				// Try a smaller step size and see if that finds a better
				// solution.
				pixelstep /= (lattice-1);			
			}

		}

		// Synchronize all of the threads in the block.
		__syncthreads();

	} while (!done);	  
}

// Optimize the passed-in list of symmetric XY trackers based on the
// image buffer passed in.
bool VST_cuda_optimize_symmetric_trackers(const VST_cuda_image_buffer &buf,
                                                 std::list<Spot_Information *> &tkrs,
                                                 unsigned num_to_optimize)
{
	// Make sure we can initialize CUDA.  This also allocates the global
	// input buffer that we'll use to copy data from the host into.
	if (!VST_ensure_cuda_ready(buf)) { return false; }
	
	// Copy the input image from host memory into the GPU buffer.
	size_t imgsize = buf.nx * buf.ny * sizeof(float);
#ifdef USE_TEXTURE
	if (hipMemcpyToArray(g_cuda_fromhost_array, 0, 0, buf.buf, imgsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy array to CUDA\n");
		return false;
	}

	// Bind the texture reference to the texture after describing it.
	g_tex_ref.addressMode[0] = hipAddressModeClamp;
	g_tex_ref.addressMode[1] = hipAddressModeClamp;

	// Do linear interpolation on the result.  NOTE: The precision of the interpolator
	// is only 9 bits, so there are only 512 possible locations at which to sample
	// within a pixel in each axis; if we want better accuracy than 1/256, we won't
	// be happy with this result.
	g_tex_ref.filterMode = hipFilterModeLinear;
	g_tex_ref.normalized = false;
	hipBindTextureToArray(g_tex_ref, g_cuda_fromhost_array, g_channel_desc);
#else
	if (hipMemcpy(g_cuda_fromhost_buffer, buf.buf, imgsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory to CUDA\n");
		return false;
	}
#endif
	
	// Allocate an array of tracker information to pass down to the kernel.
	// with one entry per tracker we are optimizing.  This stores the tracking
	// parameters associated with each tracker along with its X and Y positions;
	// the kernel will replace the X and Y locations, which are then copied back
	// into the trackers.
	if (num_to_optimize > tkrs.size()) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Not enough tracker for request\n");
		return false;
	}
	CUDA_Tracker_Info *ti = new CUDA_Tracker_Info[tkrs.size()];
	if (ti == NULL) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Out of memory\n");
		return false;
	}
	int i;
	std::list<Spot_Information *>::iterator  loop;
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		const spot_tracker_XY *t = (*loop)->xytracker();
		ti[i].radius = static_cast<float>(t->get_radius());
		ti[i].sample_separation = static_cast<float>(t->get_sample_separation());
		ti[i].pixel_accuracy = static_cast<float>(t->get_pixel_accuracy());
		ti[i].x = static_cast<float>(t->get_x());
		ti[i].y = static_cast<float>(t->get_y());
		ti[i].lost = (*loop)->lost();
	}
	
	// Allocate a GPU buffer to store the tracker information.
	// Copy the tracker information from the host to GPU memory.
	CUDA_Tracker_Info *gpu_ti;
	size_t tkrsize = num_to_optimize * sizeof(CUDA_Tracker_Info);
	if (hipMalloc((void**)&gpu_ti, tkrsize) != hipSuccess) {
	  fprintf(stderr,"VST_cuda_optimize_symmetric_trackers(): Could not allocate memory.\n");
	  delete [] ti;
	  return false;
	}
	if (hipMemcpy(gpu_ti, ti, tkrsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory to CUDA\n");
		delete [] ti;
		return false;
	}

	// We have EXACTLY as many blocks in X as we have trackers.  We have one block
	// in Y.  We have a lattice of threads within a block that should be the
	// same number in X and Y; we test on points on that lattice and then
	// divide by that size to set a smaller lattice to check.
	g_threads.x = 8;
	g_threads.y = 8;
	g_threads.z = 1;
	g_grid.x = num_to_optimize;
	g_grid.y = 1;

	// Make sure we're not asking for too many threads, or a non-square lattice.
	if ( (g_threads.x > MAX_LATTICE) || (g_threads.y > MAX_LATTICE) ) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Lattice to large\n");
		return false;
	}
	if ( g_threads.x != g_threads.y ) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Lattice not square\n");
		return false;
	}

	// Call the CUDA kernel to do the tracking, reading from
	// the input buffer and editing the positions in place.
	// Synchronize the threads when we are done so we know that 
	// they finish before we copy the memory back.
#ifdef	USE_TEXTURE
	VST_cuda_symmetric_opt_kernel<<< g_grid, g_threads >>>(g_cuda_fromhost_array,
					buf.nx, buf.ny,
					gpu_ti, num_to_optimize);
#else
	VST_cuda_symmetric_opt_kernel<<< g_grid, g_threads >>>(g_cuda_fromhost_buffer,
					buf.nx, buf.ny,
					gpu_ti, num_to_optimize);
#endif
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not synchronize threads\n");
		return false;
	}

	// Copy the tracker info back from the GPU to the host memory.
	if (hipMemcpy(ti, gpu_ti, tkrsize, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory back to host\n");
		hipFree(gpu_ti);
		delete [] ti;
		return false;
	}
	
	// Copy the positions and fitnesses back into the trackers.
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		spot_tracker_XY *t = (*loop)->xytracker();
		t->set_location(ti[i].x, ti[i].y);
		t->set_fitness(ti[i].fitness);
	}
		
	// Free the array of tracker info on the GPU and host sides.
	hipFree(gpu_ti); gpu_ti = NULL;
	delete [] ti; ti = NULL;

	// Done!
	return true;
}

// CUDA kernel to check the fitness values in rings around the passed-in
// set of trackers, filling in the values around the ring.
// We have 1 thread in X and 512 threads in Y.
// We have as many blocks in X as we have trackers.
// We have as many blocks in Y as we have points to sample around
// the maximum-radius tracker location divided by 512 (the number of
// threads).

#ifdef	USE_TEXTURE
static __global__ void VST_cuda_symmetric_bright_lost_kernel(const hipArray *img, int nx, int ny,
							CUDA_Tracker_Info *tkrs, int nt,
							float *fitness, unsigned num_radii)
#else
static __global__ void VST_cuda_symmetric_bright_lost_kernel(const float *img, int nx, int ny,
							CUDA_Tracker_Info *tkrs, int nt,
							float *fitness, unsigned num_radii)
#endif
{
  // Figure out which tracker and which spot within that tracker we're doing.
  int tkr_id = blockIdx.x;
  int spot_id = blockIdx.y * blockDim.y + threadIdx.y;
  
  // Figure out our offset from our tracker base, and where we should store our
  // result in the fitness array.  This calculation is redundant with that done
  // on the host, but it seems likely to be faster to recompute it here than to
  // try and fill in all the info we need and pass it down.
  unsigned r;					// Goes through the indices of radii
  unsigned point_count;			// How many points found for this radius
  unsigned total_count = 0;		// How many total points found so far
  float my_radius = -1;			// Pixel distance from tracker center
  int my_theta_index = -1;		// Index of my location around the circumference
  for (r = 0; r < num_radii; r++) {
	float radius = 2*r + 3;		// Actual radius we're checking.
	point_count = static_cast<unsigned>( 2 * HIP_PI_F * radius );
	if (total_count + point_count > spot_id) {
		my_radius = r;
		my_theta_index = spot_id - total_count;
	}
	total_count += point_count;
  }
  // Figure out where I should put my answer.
  unsigned my_fitness_index = tkr_id * total_count + spot_id;
  
  // If we're a valid tracker and a valid spot_id, then compute the fitness for
  // our tracker at the specified offset and store it into the fitness array.
  if ( (tkr_id < nt) && (my_radius > 0) ) {
	// Make my own tracker with its information copied from the original
	// passed-in tracker I'm associated with.
	CUDA_Tracker_Info member_t = tkrs[tkr_id];

	// Compute the offset from the original location, then set it to that
	// offset.
	float theta = (1.0f/my_radius) * my_theta_index;
	float x = member_t.x + my_radius * cos(theta);
	float y = member_t.y + my_radius * sin(theta);
	member_t.x = x;
	member_t.y = y;
	
	fitness[my_fitness_index] = cuda_check_fitness_symmetric(img, nx, ny, member_t); 
  }
}

// Optimize the passed-in list of symmetric XY trackers based on the
// image buffer passed in.
bool VST_cuda_check_bright_lost_symmetric_trackers(const VST_cuda_image_buffer &buf,
                                                 std::list<Spot_Information *> &tkrs,
                                                 unsigned num_to_optimize,
                                                 float var_thresh)
{
	// Make sure we can initialize CUDA.  This also allocates the global
	// input buffer that we'll use to copy data from the host into.
	if (!VST_ensure_cuda_ready(buf)) { return false; }
	
	// Copy the input image from host memory into the GPU buffer.
	size_t imgsize = buf.nx * buf.ny * sizeof(float);
#ifdef USE_TEXTURE
	if (hipMemcpyToArray(g_cuda_fromhost_array, 0, 0, buf.buf, imgsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_check_bright_lost_symmetric_trackers(): Could not copy array to CUDA\n");
		return false;
	}

	// Bind the texture reference to the texture after describing it.
	g_tex_ref.addressMode[0] = hipAddressModeClamp;
	g_tex_ref.addressMode[1] = hipAddressModeClamp;

	// Do linear interpolation on the result.  NOTE: The precision of the interpolator
	// is only 9 bits, so there are only 512 possible locations at which to sample
	// within a pixel in each axis; if we want better accuracy than 1/256, we won't
	// be happy with this result.
	g_tex_ref.filterMode = hipFilterModeLinear;
	g_tex_ref.normalized = false;
	hipBindTextureToArray(g_tex_ref, g_cuda_fromhost_array, g_channel_desc);
#else
	if (hipMemcpy(g_cuda_fromhost_buffer, buf.buf, imgsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_check_bright_lost_symmetric_trackers(): Could not copy memory to CUDA\n");
		return false;
	}
#endif
	
	// Allocate an array of tracker information to pass down to the kernel.
	// with one entry per tracker we are checking.  This stores the tracking
	// parameters associated with each tracker along with its X and Y positions;
	// the kernel will replace the 'lost' value, which are then copied back
	// into the trackers.
	CUDA_Tracker_Info *ti = new CUDA_Tracker_Info[tkrs.size()];
	if (ti == NULL) {
		fprintf(stderr, "VST_cuda_check_bright_lost_symmetric_trackers(): Out of memory\n");
		return false;
	}
	int i;
	std::list<Spot_Information *>::iterator  loop;
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		spot_tracker_XY *t = (*loop)->xytracker();
		ti[i].radius = static_cast<float>(t->get_radius());
		ti[i].sample_separation = static_cast<float>(t->get_sample_separation());
		ti[i].pixel_accuracy = static_cast<float>(t->get_pixel_accuracy());
		ti[i].x = static_cast<float>(t->get_x());
		ti[i].y = static_cast<float>(t->get_y());
		ti[i].lost = (*loop)->lost();
	}
	
	// Allocate a GPU buffer to store the tracker information.
	// Copy the tracker information from the host to GPU memory.
	CUDA_Tracker_Info *gpu_ti;
	size_t tkrsize = num_to_optimize * sizeof(CUDA_Tracker_Info);
	if (hipMalloc((void**)&gpu_ti, tkrsize) != hipSuccess) {
	  fprintf(stderr,"VST_cuda_check_bright_lost_symmetric_trackers(): Could not allocate memory.\n");
	  delete [] ti;
	  return false;
	}
	if (hipMemcpy(gpu_ti, ti, tkrsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_check_bright_lost_symmetric_trackers(): Could not copy memory to CUDA\n");
		delete [] ti;
		return false;
	}
	
	// Figure out how many samples we need to take in the worst case for
	// any tracker in our set.  This depends on the maximum radius among
	// the trackers -- we take a sample every pixel around rings, and the
	// rings go in steps of 2 pixels out from the center, starting at
	// a radius of 3 pixels.  Also figure out the start index into an
	// array of values for each ring.
	double max_radius = 0;
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		double radius = ti[i].radius;
		if (radius > max_radius) {
			max_radius = radius;
		}
	}
	int num_radii = static_cast<int>( (max_radius - 3)/2 + 1 );
	unsigned *point_counts = new unsigned[num_radii];
	unsigned *start_index = new unsigned[num_radii];
	if (!point_counts || !start_index) {
	  fprintf(stderr,"VST_cuda_check_bright_lost_symmetric_trackers(): Could not allocate index memory.\n");
	  delete [] ti;
	  return false;
	}
	unsigned total_points = 0;
	for (i = 0; i < num_radii; i++) {
		double radius = 2*i + 3;
		point_counts[i] = static_cast<unsigned>( 2 * M_PI * radius );
		total_points += point_counts[i];
		start_index[i] = total_points - point_counts[i];
	}
	
	// Allocate the array of fitness values with as many points per
	// tracker as there are in the maximum tracker.  We don't fill
	// these in here -- they will be filled in on the GPU.
	float *gpu_fitness;
	size_t fitness_size = num_to_optimize * total_points * sizeof(float);
	if (hipMalloc((void**)&gpu_fitness, fitness_size) != hipSuccess) {
	  fprintf(stderr,"VST_cuda_check_bright_lost_symmetric_trackers(): Could not allocate fitness memory.\n");
	  delete [] start_index;
	  delete [] point_counts;
	  delete [] ti;
	  return false;
	}
	
    // We have 1 thread in X and 512 threads in Y.
    // We have as many blocks in X as we have trackers.
    // We have as many blocks in Y as we have points to sample around
    // the maximum-radius tracker location divided by 512 (the number of
    // threads).
    g_threads.x = 1;
    g_threads.y = 512;
    g_threads.z = 1;
    g_grid.x = num_to_optimize;
    g_grid.y = (total_points / 512) + 1;
	
	// Call the CUDA kernel to compute the fitness values at.
	// the proper locations around the start for each tracker.
	// Synchronize the threads when we are done so we know that 
	// they finish before we copy the memory back.
#ifdef	USE_TEXTURE
	VST_cuda_symmetric_bright_lost_kernel<<< g_grid, g_threads >>>(g_cuda_fromhost_array,
					buf.nx, buf.ny,
					gpu_ti, num_to_optimize,
					gpu_fitness, num_radii);
#else
	VST_cuda_symmetric_bright_lost_kernel<<< g_grid, g_threads >>>(g_cuda_fromhost_buffer,
					buf.nx, buf.ny,
					gpu_ti, num_to_optimize,
					gpu_fitness, num_radii);
#endif
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "VST_cuda_check_bright_lost_symmetric_trackers(): Could not synchronize threads\n");
		return false;
	}

	// Allocate a buffer to store the host-side fitness values.
	// Copy the fitness info back from the GPU to the host memory.
	float *fitness = new float[fitness_size / sizeof(float)];
	if (!fitness) {
		fprintf(stderr, "VST_cuda_check_bright_lost_symmetric_trackers(): Could not allocate host fitness memory\n");
		hipFree(gpu_fitness);
		hipFree(gpu_ti);
		delete [] start_index;
		delete [] point_counts;
		delete [] ti;
		return false;
	}
	if (hipMemcpy(fitness, gpu_fitness, fitness_size, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "VST_cuda_check_bright_lost_symmetric_trackers(): Could not copy memory back to host\n");
		hipFree(gpu_fitness);
		hipFree(gpu_ti);
		delete [] fitness;
		delete [] start_index;
		delete [] point_counts;
		delete [] ti;
		return false;
	}
	
	// Determine whether each tracker is lost by looking at the minimum of
	// the maxima around the rings and comparing it to the lost-tracking
	// parameter.
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		float min_val = 1e20f;
		int r;
		num_radii = static_cast<int>( ((*loop)->xytracker()->get_radius() - 3)/2 + 1 );
		for (r = 0; r < num_radii; r++) {
			float max_val = -1e20f;
			unsigned j;
			for (j = 0; j < point_counts[r]; j++) {
				float val = fitness[ i*total_points + start_index[r] + j ];
				if (val > max_val) {
					max_val = val;
				}
			}
			if (max_val < min_val) {
				min_val = max_val;
			}
		}
		double scale_factor = 1 + 9*var_thresh;
		bool am_lost = false;
        if ((*loop)->xytracker()->get_fitness() * scale_factor < min_val) {
			am_lost = true;
		}
		(*loop)->lost(am_lost);
	}
		
	// Free the array of tracker info on the GPU and host sides.
	hipFree(gpu_fitness); gpu_fitness = NULL;
	hipFree(gpu_ti); gpu_ti = NULL;
	delete [] fitness;
	delete [] start_index;
	delete [] point_counts;
	delete [] ti; ti = NULL;

	// Done!
	return true;
}


//----------------------------------------------------------------------
// Notes on speedup attempts for tracking are below here
//----------------------------------------------------------------------

/*
Speed with CPU on two 20-radius trackers with precision 0.05 and spacing 0.5: 55 fps.
Speed with initial GPU implementation: 3.8 fps.
Changing the radius to 10 makes it12.7-13.2 fps.
Changing the sample separation from 0.5 to 1 for radius 20 makes it 7.1 fps.
Changing the sample separation from 0.5 to 0.1 for radius 20 makes it 0.8 fps.
Changing the precision from 0.05 to 0.5 makes it 8.1 fps.

Switching to using the bilinear interpolation texture hardware makes it around 7 fps.
	(turning off if() checks within bilerp only increased to 8.6 fps)
	(calling synchthreads has no impact -- only one thread per block!)
Running 32 threads in parallel per tracker in a fan pattern around the origin got 29 fps.
	This is 60 fps for a 10-pixel tracker.  And it scales up to lots of trackers.
	Changing sample spacing to 1.0 gets us up to 43 fps.
Running a lattice of 32x32 points around the current location got us to 24 fps.
Running a lattice of 24x24 points around the current location got us to 35 fps.
Running a lattice of 16x16 points around the current location got us to 39 fps.
	Slowed down to 36 with 8 trackers.
Running a lattice of 12x12 points around the current location got us to 39 fps.
Running a lattice of 8x8 points around the current location got us 38 fps.
Running a lattice of 6x6 points around the current location got us to 32 fps.
Running a lattice of 4x4 points around the current location got us to 33 fps.
	Stayed the same speed for 8 trackers.
Running a lattice of 2x2 points around the current location got us to 21 fps.
	Just checking the scaling, didn't expect speedup.

Recomputing the "are we done" sooner at a lattice of 4x4 points gets us to 38

Switching from cos() and sin() to sincos() gets us to 44 fps.

Switching the count from int to float and re-ordering the texure read in
	cuda_read_pixel_bilerp gets us up to 50 fps.
	
Pulling the bilerp texture call into the code from inline gets us to 52 fps.

*** Bug found.  The pixel separation around the circle on the symmetric
    tracker was being computed incorrectly.  This resulted in too few
    points being sampled.  This was in both the serial and the CUDA code.
    Timings below are from the new code.
    
Serial code ran at 34.5 fps.
CUDA code ran at pretty much the exact same speed.  It slowed down a little with
		9 beads.
Switching to only computing half the sin() and cos() and re-using them got 43
After fixing some bugs and passing information back, it got to 49
	This is surprising -- it should have gone slower.
	
Ideas:
	Precompute the kernel offsets and store them in shared memory
			Taking out the sin() and cos() from the inner loop speeds up to 60
	Remove if statements in inner loop (bilerp) (got a little less than 2x).
		Texture version is set to clamp, but this is not quite what we want.
		Pad image with -1 around border and use that to squash?
*/

#include "hip/hip_runtime.h"
/*********************************************************************
This is the source file for the CUDA version of the spot_tracker library
function calls..

WARNING: All of the CUDA code for the entire project has to be in here
so that we only initialize the device once.
**********************************************************************/

#include "image_wrapper.h"
#include "spot_tracker.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

//----------------------------------------------------------------------
// Definitions and routines needed by all functions below.
//----------------------------------------------------------------------

static hipDevice_t     g_cuDevice;     // CUDA device
static hipCtx_t    g_cuContext;    // CUDA context on the device
static float        *g_cuda_fromhost_buffer = NULL;
static unsigned		g_cuda_fromhost_nx = 0;
static unsigned		g_cuda_fromhost_ny = 0;

// For the GPU code, block size and number of kernels to run to cover a whole grid.
// Initialized once in VST_ensure_cuda_ready();
static dim3         g_threads;      // 16x16x1
static dim3         g_grid;         // Computed to cover array (slightly larger than array)

// Open the CUDA device and get a context.  Also allocate a buffer of
// appropriate size.  Do this allocation only when the size of the buffer
// allocated is different from the newly-requested size.  Return false
// if we cannot get one.  This function can be called every time a
// CUDA_using function is called, but it only does the device opening
// and image-buffer allocation once.
static bool VST_ensure_cuda_ready(const VST_cuda_image_buffer &inbuf)
{
  static bool initialized = false;	// Have we initialized yet?
  static bool okay = false;			// Did the initialization work?
  if (!initialized) {
    // Whether this works or not, we'll be initialized.
    initialized = true;
    
    // Open the largest-ID CUDA device in the system
    hipInit(0);
    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    if (num_devices == 0) {
      fprintf(stderr,"VST_ensure_cuda_ready(): No CUDA devices.\n");
      return false;
    }
    if (hipDeviceGet(&g_cuDevice, num_devices-1) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get device.\n");
      return false;
    }
    if (hipCtxCreate( &g_cuContext, 0, g_cuDevice ) != hipSuccess) {
      fprintf(stderr,"VST_ensure_cuda_ready(): Could not get context.\n");
      return false;
    }
    
    // Allocate a buffer to be used on the GPU.  It will be
    // copied from host memory.
    if ( (inbuf.nx != g_cuda_fromhost_nx) || (inbuf.ny != g_cuda_fromhost_nx) ) {
	    
		unsigned int numBytes = inbuf.nx * inbuf.ny * sizeof(float);
		if (g_cuda_fromhost_buffer != NULL) {
			hipFree(g_cuda_fromhost_buffer);
		}
		if (hipMalloc((void**)&g_cuda_fromhost_buffer, numBytes) != hipSuccess) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Could not allocate memory.\n");
		  return false;
		}
		if (g_cuda_fromhost_buffer == NULL) {
		  fprintf(stderr,"VST_ensure_cuda_ready(): Buffer is NULL pointer.\n");
		  return false;
		}
		g_cuda_fromhost_nx = inbuf.nx;
		g_cuda_fromhost_ny = inbuf.ny;
	}
	
    // Set up enough threads (and enough blocks of threads) to at least
    // cover the size of the array.  We use a thread block size of 16x16
    // because that's what the example matrixMul code from nVidia does.
    // Changing them to 8 and 8 makes the Gaussian kernel slower.  Changing
    // them to 32 and 32 also makes it slower (by not as much)
    g_threads.x = 16;
    g_threads.y = 16;
    g_threads.z = 1;
    g_grid.x = (g_cuda_fromhost_nx / g_threads.x) + 1;
    g_grid.y = (g_cuda_fromhost_ny / g_threads.y) + 1;
    g_grid.z = 1;	

    // Everything worked, so we're okay.
    okay = true;
  }

  // Return true if we are okay.
  return okay;
}

//----------------------------------------------------------------------
// Functions called from image_wrapper.cpp.
//----------------------------------------------------------------------

// Compute the value of a Gaussian at the specified point.  The function is 2D,
// centered at the origin.  The "standard normal distribution" Gaussian has an integrated
// volume of 1 over all space and a variance of 1.  It is defined as:
//               1           -(R^2)/2
//   G(x) = ------------ * e
//             2*PI
// where R is the radius of the sample point from the origin.
// We let the user set the standard deviation s, changing the function to:
//                  1           -(R^2)/(2*s^2)
//   G(x) = --------------- * e
//           s^2 * 2*PI
// For computational efficiency, we refactor this into A * e ^ (B * R^2).

inline __device__ float	cuda_Gaussian(
  const float &s_meters,      //< standard deviation (square root of variance)
  const float &x, const float &y)	//< Point to sample (relative to origin)
{
  float variance = s_meters * s_meters;
  float R_squared = x*x + y*y;

  const float twoPI = static_cast<float>(2*HIP_PI_F);
  const float twoPIinv = 1.0f / twoPI;
  float B = -1 / (2 * variance);
  float A = twoPIinv / variance;

  return A * __expf(B * R_squared);
}

// CUDA kernel to do a Gaussian blur of the passed-in image and place
// it into the output images.
// Told the buffer beginning and the buffer size.  Assumes at least
// as many threads are run as there are elements in the buffer.
// Assumes a 2D array of threads.
static __global__ void VST_cuda_gaussian_blur(float *in, float *out, int nx, int ny,
							unsigned aperture, float std)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( (x < nx) && (y < ny) ) {

	// Replacing the cuda_Gaussian() calls below with 1 changed the speed from
	// 23 frames/second to 25 frames/second, so it is not the bottleneck.
	// Replacing the in[] read call with 1 slows things down to 21.
	// Replacing the inside-if with (weight++; sum++) speeds things to 29.
	// Removing the if() test speeds things up to 45 (but gets funky answers).
	// Pulling the if() statement out of the inner loop and into the bounds
	// setting for the i and j loops made the speed 39.
	// Swapping the sum += and weight += lines below to put weight later brought
	// us up to 43.
	// Swapping the kval = and value = lines to put kval first brought it up
	// to 43.8.
	// Moving the kval and value definitions outside the loop dropped back to
	// 39.
	// Moving the definition of int j into the i loop bumped it up to 44.  Looks
	// like the compiler doesn't always do the best optimizing for us...
	// After the above mods, changing the cuda_Gaussian() to = 1.0f made things
	// go 53 frames/second, so there may be some computational gain to be had
	// in there.
	// XXX Switching the code below to one like the faster algorithm in the
	// CPU code may speed things up a bit more.
	// Changing the radius (affects the aperture) from 5 to 3 makes things
	// go 41 frames/second.
    // If we don't have an integer version of aperture, the "-aperture"
    // below turns into a large positive number, meaning that
    // the loops never get executed.
    int aperture_int = aperture;
    
    // Determine the safe bounds to read from around this point.  This avoids
    // having to put an if() statement in the inner loop, which slows us down
    // a bunch.
    int min_i = -aperture_int;
    int max_i = aperture_int;
    int min_j = -aperture_int;
    int max_j = aperture_int;
    int min_x = x - aperture_int; if (min_x < 0) { min_i = -min_x; }
    int min_y = y - aperture_int; if (min_y < 0) { min_j = -min_y; }
    int max_x = x + aperture_int; if (max_x >= nx) { max_i -= ( max_x - (nx-1) ); }
    int max_y = y + aperture_int; if (max_y >= ny) { max_j -= ( max_y - (ny-1) ); }
    int i;
    float sum = 0;
    float weight = 0;
    for (i = min_i; i <= max_i; i++) {
	  int j;
      for (j = min_j; j <= max_j; j++) {
          float kval = cuda_Gaussian(std,i,j);
		  float value = in[x+i + (y+j)*nx];
          sum += kval * value;
          weight += kval;
      }
    }
	out[x + y*nx] = sum/weight;
	
  }
}

bool VST_cuda_blur_image(VST_cuda_image_buffer &buf, unsigned aperture, float std)
{
	// Make sure we can initialize CUDA.  This also allocates the global
	// input buffer that we'll copy data from the host into.
	if (!VST_ensure_cuda_ready(buf)) { return false; }
	
	// Copy the input image from host memory into the GPU buffer.
	size_t size = buf.nx * buf.ny * sizeof(float);
	if (hipMemcpy(g_cuda_fromhost_buffer, buf.buf, size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not copy memory to CUDA\n");
		return false;
	}
	
	// Allocate a CUDA buffer to blur into from the input buffer.  It should
	// be the same size as the input buffer.  We only allocate this when the
	// size changes.
	static int blur_nx = 0;
	static int blur_ny = 0;
	static float *blur_buf = NULL;
	if ( (blur_nx != g_cuda_fromhost_nx) || (blur_ny != g_cuda_fromhost_ny) ) {
		if (blur_buf != NULL) { hipFree(blur_buf); }
		blur_nx = g_cuda_fromhost_nx;
		blur_ny = g_cuda_fromhost_ny;
		if (hipMalloc((void**)&blur_buf, size) != hipSuccess) {
		  fprintf(stderr,"VST_cuda_blur_image(): Could not allocate memory.\n");
		  return false;
		}
	}
	if (blur_buf == NULL) {
	  fprintf(stderr,"VST_cuda_blur_image(): Buffer is NULL pointer.\n");
	  return false;
	}
	
	// Call the CUDA kernel to do the blurring on the image, reading from
	// the global input buffer and writing to the blur buffer.
	// Synchronize the threads when
	// we are done so we know that they finish before we copy the memory
	// back.
	VST_cuda_gaussian_blur<<< g_grid, g_threads >>>(g_cuda_fromhost_buffer,
					blur_buf, blur_nx, blur_ny, aperture, std);
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not synchronize threads\n");
		return false;
	}

	// Copy the buffer back from the GPU to host memory.
	if (hipMemcpy(buf.buf, blur_buf, size, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not copy memory back to host\n");
		return false;
	}
	
	// Everything worked!
	return true;
}

//----------------------------------------------------------------------
// Functions called from spot_tracker.cpp.
//----------------------------------------------------------------------

typedef struct {
	float radius;
	float sample_separation;
	float pixel_accuracy;
	float x;
	float y;
	float fitness;
	float pixelstep;
} CUDA_Tracker_Info;

// Find the maximum of three elements.  Return the
// index of which one was picked.
inline __device__ float max3(const float &v0, const float &v1, const float &v2,
							 unsigned &index) {
  float max = v0; index = 0;
  if (v1 > max) { max = v1; index = 1; }
  if (v2 > max) { max = v2; index = 2; }
  return max;
}

// Read a pixel value from the specified image.
// Return false if the coordinate its outside the image.
// Return the correct result and true if the coordinate is inside.
inline __device__ bool cuda_read_pixel(
	const float *img, const int &nx, const int &ny,
	const int &x, const int &y,
	float &result)
{
	if ( (x < 0) || (y < 0) || (x >= nx) || (y >= ny) ) {
		return false;
	}
	result = img[x + y*nx];
	return true;
}

// Do bilinear interpolation to read from the image, in order to
// smoothly interpolate between pixel values.
// All sorts of speed tweaks in here because it is in the inner loop for
// the spot tracker and other codes.
// Return a result of zero and false if the coordinate its outside the image.
// Return the correct interpolated result and true if the coordinate is inside.
// XXX Later, consider using the hardware-accelerated bilerp function by using
// a texture to read from; beware that the interpolator will only do a maximum
// of 256 subpixel locations, which won't get us below a resolution of 1/256.
inline __device__ bool cuda_read_pixel_bilerp(
	const float *img, const int &nx, const int &ny,
	const float &x, const float &y,
	float &result)
{
	result = 0;	// In case of failure.
	// The order of the following statements is optimized for speed.
	// The float version is used below for xlowfrac comp, ixlow also used later.
	// Slightly faster to explicitly compute both here to keep the answer around.
	float xlow = floor(x); int ixlow = (int)xlow;
	// The float version is used below for ylowfrac comp, ixlow also used later
	// Slightly faster to explicitly compute both here to keep the answer around.
	float ylow = floor(y); int iylow = (int)ylow;
	int ixhigh = ixlow+1;
	int iyhigh = iylow+1;
	float xhighfrac = x - xlow;
	float yhighfrac = y - ylow;
	float xlowfrac = 1.0 - xhighfrac;
	float ylowfrac = 1.0 - yhighfrac;
	float ll, lh, hl, hh;

	// Combining the if statements into one using || makes it slightly slower.
	// Interleaving the result calculation with the returns makes it slower.
	if (!cuda_read_pixel(img, nx, ny, ixlow, iylow, ll)) { return false; }
	if (!cuda_read_pixel(img, nx, ny, ixlow, iyhigh, lh)) { return false; }
	if (!cuda_read_pixel(img, nx, ny, ixhigh, iylow, hl)) { return false; }
	if (!cuda_read_pixel(img, nx, ny, ixhigh, iyhigh, hh)) { return false; }
	result = ll * xlowfrac * ylowfrac + 
		 lh * xlowfrac * yhighfrac +
		 hl * xhighfrac * ylowfrac +
		 hh * xhighfrac * yhighfrac;
	return true;
};

// Check the fitness of the specified symmetric tracker within the
// specified image at the specified location.
// This code should compute exactly the same thing as the
// symmetric_spot_tracker_interp function.
// XXX Later, put the locations to search into constant memory rather
// than computing them on the fly here.
inline __device__ float	cuda_check_fitness_symmetric(
	const float *img, const int &nx, const int &ny,
	const CUDA_Tracker_Info &tkr)
{
	// Construct aliases for the parameters that give us easy local names.
	const float &radius = tkr.radius;
	const float &samplesep = tkr.sample_separation;
	const float &x = tkr.x;
	const float &y = tkr.y;
	
	// Sum up over rings that are samplesep away from the center; we
	// don't count the center pixel because it will never have variance.
	float r;
	float ring_variance_sum = 0;
	for (r = samplesep; r <= radius; r += samplesep) {
		int count = 0;
		float valSum = 0.0f;
		float squareValSum = 0.0f;
		float rads_per_step = 1.0f / r;
		float start = (r/samplesep)*rads_per_step*0.5f;
		float theta;
		for (theta = start; theta < 2*HIP_PI_F + start; theta += rads_per_step) {
			float newx = x + r*cos(theta);
			float newy = y + r*sin(theta);
			float val;
			if (cuda_read_pixel_bilerp(img, nx, ny, newx, newy, val)) {
				squareValSum += val*val;
				valSum += val;
				count++;
			}
		}
	
		if (count) {
			ring_variance_sum += squareValSum - valSum*valSum / count;
		}
	}
	
	return -ring_variance_sum;
}

// Optimize starting at the specified location to find the best-fit disk.
// Take only one optimization step.  Return whether we ended up finding a
// better location or not.  Return new location in any case.  One step means
// one step in X,Y, and radius space each.  We always optimize in xy only.
inline __device__ bool cuda_take_single_optimization_step_xy(const float *img,
							const int &nx, const int &ny,
							CUDA_Tracker_Info &t)
{
  // Aliases to make it easier to refer to things.
  const float &pixelstep = t.pixelstep;
  float &fitness = t.fitness;
  float &x = t.x;
  float &y = t.y;
  
  float	  new_fitness;	    //< Checked fitness value to see if it is better than current one
  bool	  betterxy = false; //< Do we find a better location?

  // Try going in +/- X and see if we find a better location.  It is
  // important that we check both directions before deciding to step
  // to avoid unbiased estimations.
  {
    float v0, vplus, vminus;
    float starting_x = x;
    v0 = fitness;                                   // Value at starting location
    // XXX Need to do this differently when we run in parallel.  Also for Y.
    x = starting_x + pixelstep;
    vplus = cuda_check_fitness_symmetric(img, nx, ny, t);
    x = starting_x - pixelstep;
    vminus = cuda_check_fitness_symmetric(img, nx, ny, t);
    unsigned which;
    new_fitness = max3(v0, vplus, vminus, which);
    switch (which) {
      case 0: x = starting_x;
              break;
      case 1: x = starting_x + pixelstep;
              betterxy = true;
              break;
      case 2: x = starting_x - pixelstep;
              betterxy = true;
              break;
    }
    fitness = new_fitness;
  }
  
  // Try going in +/- Y and see if we find a better location.  It is
  // important that we check both directions before deciding to step
  // to avoid unbiased estimations.
  {
    float v0, vplus, vminus;
    float starting_y = y;
    v0 = fitness;                                      // Value at starting location
    y = starting_y + pixelstep;	// Try going a step in +Y
    vplus = cuda_check_fitness_symmetric(img, nx, ny, t);
    y = starting_y - pixelstep;	// Try going a step in +Y
    vminus = cuda_check_fitness_symmetric(img, nx, ny, t);
    unsigned which;
    new_fitness = max3(v0, vplus, vminus, which);
    switch (which) {
      case 0: y = starting_y;
              break;
      case 1: y = starting_y + pixelstep;
              betterxy = true;
              break;
      case 2: y = starting_y - pixelstep;
              betterxy = true;
              break;
    }
    fitness = new_fitness;
  }
  
  // Tell if we found a better location.
  return betterxy;
}

// CUDA kernel to optimize the passed-in list of trackers based on the
// passed-in image.  Moves the X and Y position of each tracker to its
// final optimum location.
// Assumes that we have at least as many threads in X as we have trackers.
// Assumes a 2D array of threads.
// XXX Later, at least use four threads for the four directional checks
// at each level to speed things up.
static __global__ void VST_cuda_symmetric_opt_kernel(const float *img, int nx, int ny,
							CUDA_Tracker_Info *tkrs, int nt)
{
  // For now, just do one thread per tracker and have it be the one with y=0.
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;
  int tkr_id = tx;
  int group_id = ty;

  // Do the whole optimization here, checking with smaller and smaller
  // steps until we reach the accuracy requested.
  if ( (tkr_id < nt) && (group_id == 0) ) {
  
	// Get local aliases for the passed-in variables we need to use.    
	CUDA_Tracker_Info &t = tkrs[tkr_id];
	const float &accuracy = t.pixel_accuracy;
	float &fitness = t.fitness;	// Keeps track of current fitness
	float &pixelstep = t.pixelstep;
	pixelstep = 2.0f;	// Start with a large value.
	
	fitness = cuda_check_fitness_symmetric(img, nx, ny, t);
	do {
		while (cuda_take_single_optimization_step_xy(img, nx, ny, t)) {};
		if (pixelstep <= accuracy) {
			break;
		}
		pixelstep /= 2.0f;
	} while (true);
	
  }
}

// Optimize the passed-in list of symmetric XY trackers based on the
bool VST_cuda_optimize_symmetric_trackers(const VST_cuda_image_buffer &buf,
                                                 std::list<Spot_Information *> &tkrs,
                                                 unsigned num_to_optimize)
{
	// Make sure we can initialize CUDA.  This also allocates the global
	// input buffer that we'll copy data from the host into.
	if (!VST_ensure_cuda_ready(buf)) { return false; }
	
	// Copy the input image from host memory into the GPU buffer.
	size_t imgsize = buf.nx * buf.ny * sizeof(float);
	if (hipMemcpy(g_cuda_fromhost_buffer, buf.buf, imgsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory to CUDA\n");
		return false;
	}
	
	// Allocate an array of tracker information to pass down to the kernel.
	// with one entry per tracker we are optimizing.  This stores the tracking
	// parameters associated with each tracker along with its X and Y positions;
	// the kernel will replace the X and Y locations, which are then copied back
	// into the trackers.
	CUDA_Tracker_Info *ti = new CUDA_Tracker_Info[tkrs.size()];
	if (ti == NULL) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Out of memory\n");
		return false;
	}
	int i;
	std::list<Spot_Information *>::iterator  loop;
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		spot_tracker_XY *t = (*loop)->xytracker();
		ti[i].radius = static_cast<float>(t->get_radius());
		ti[i].sample_separation = static_cast<float>(t->get_sample_separation());
		ti[i].pixel_accuracy = static_cast<float>(t->get_pixel_accuracy());
		ti[i].x = static_cast<float>(t->get_x());
		ti[i].y = static_cast<float>(t->get_y());
	}
	
	// Allocate a GPU buffer to store the tracker information.
	// Copy the tracker information from the host to GPU memory.
	CUDA_Tracker_Info *gpu_ti;
	size_t tkrsize = num_to_optimize * sizeof(CUDA_Tracker_Info);
	if (hipMalloc((void**)&gpu_ti, tkrsize) != hipSuccess) {
	  fprintf(stderr,"VST_cuda_optimize_symmetric_trackers(): Could not allocate memory.\n");
	  delete [] ti;
	  return false;
	}
	if (hipMemcpy(gpu_ti, ti, tkrsize, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory to CUDA\n");
		delete [] ti;
		return false;
	}
	
    // Set up enough threads (and enough blocks of threads) to at least
    // cover the number of trackers in X, and we have only one thread per
    // tracker (so y = 1).  We use a thread block size of 1x1
    // because we don't want to share a cache between different trackers.
	// XXX Later, consider how to make use of multiple threads in the
	// same tracker (doing 4+ offsets at once).
    g_threads.x = 1;
    g_threads.y = 1;
    g_threads.z = 1;
    g_grid.x = num_to_optimize;
    g_grid.y = 1;
    g_grid.z = 1;	
	
	// Call the CUDA kernel to do the tracking, reading from
	// the input buffer and editing the positions in place.
	// Synchronize the threads when we are done so we know that 
	// they finish before we copy the memory back.
	VST_cuda_symmetric_opt_kernel<<< g_grid, g_threads >>>(g_cuda_fromhost_buffer,
					buf.nx, buf.ny,
					gpu_ti, num_to_optimize);
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "VST_cuda_blur_image(): Could not synchronize threads\n");
		return false;
	}

	// Copy the tracker info back from the GPU to the host memory.
	if (hipMemcpy(ti, gpu_ti, tkrsize, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "VST_cuda_optimize_symmetric_trackers(): Could not copy memory back to host\n");
		hipFree(gpu_ti);
		delete [] ti;
		return false;
	}
	
	// Copy the positions back into the trackers.
	for (loop = tkrs.begin(), i = 0; i < (int)(num_to_optimize); loop++, i++) {
		spot_tracker_XY *t = (*loop)->xytracker();
		t->set_location(ti[i].x, ti[i].y);
	}
		
	// Free the array of tracker info on the GPU and host sides.
	hipFree(gpu_ti); gpu_ti = NULL;
	delete [] ti; ti = NULL;

	// Done!
	return true;
}
